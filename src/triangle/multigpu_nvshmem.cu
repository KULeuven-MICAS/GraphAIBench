#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include "dist.h"
#include "utils.h"
#include "graph_gpu.h"
#include "graph_partition.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_vertex_nvshmem.cuh"

void TCSolver(Graph &g, uint64_t &total, int n_partitions, int chunk_size) {
#ifdef USE_MPI
  int rank = 0, size = 1;
  initialize_mpi(rank, size);
  if (rank == 0) print_device_info(0);
#endif
  auto nv = g.V();
  auto md = g.get_max_degree();
  auto infile = g.get_inputfile_prefix()+"-part"+std::to_string(rank);
  int ndevices = 0;
  CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
  if (ndevices < n_partitions) {
    std::cout << "Only " << ndevices << " GPUs available\n";
    exit(1);
  } else ndevices = n_partitions;
  int subgraph_size = (nv-1) / n_partitions + 1;
  if (rank == 0) std::cout << "subgraph size: " << subgraph_size << "\n";

  eidType max_subg_ne = 0;
#ifdef USE_MPI
  g.deallocate();
  Graph subg(infile);
  eidType subg_ne = subg.E();
  MPI_Allreduce(&subg_ne, &max_subg_ne, 1, MPI_INT64_T, MPI_MAX, MPI_COMM_WORLD);
  if (rank == 0) std::cout << "Maximum edge count in the subgraphs: " << max_subg_ne << "\n";
#else
  PartitionedGraph pg(&g, ndevices);
  pg.edgecut_partition1D();
  auto num_subgraphs = pg.get_num_subgraphs();
  assert(num_subgraphs == n_partitions);
  for (int i = 0; i < ndevices; i++) {
    auto subg_ne = pg.get_subgraph(i)->E();
    if (subg_ne > max_subg_ne) 
      max_subg_ne = subg_ne;
  }
#endif

#ifdef USE_MPI
  int local_rank = -1;
  int local_size = 1;
  {
    MPI_Comm local_comm;
    MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL, &local_comm));
    MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
    MPI_CALL(MPI_Comm_size(local_comm, &local_size));
    MPI_CALL(MPI_Comm_free(&local_comm));
  }
  if ( 1 < ndevices && ndevices < local_size ) {
    fprintf(stderr,"ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n", ndevices, local_size);
    MPI_CALL(MPI_Finalize());
    exit(1);
  }
  if (1 == ndevices) {
    // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
    CUDA_SAFE_CALL(hipSetDevice(0));
  } else {
    CUDA_SAFE_CALL(hipSetDevice(local_rank));
  }
  CUDA_SAFE_CALL(hipFree(0));
  MPI_Comm mpi_comm;
  nvshmemx_init_attr_t attr;
  mpi_comm = MPI_COMM_WORLD;
  attr.mpi_comm = &mpi_comm;
  long long unsigned required_symmetric_heap_size = (nv+1) * sizeof(eidType) + max_subg_ne * sizeof(vidType);
  char * value = getenv("NVSHMEM_SYMMETRIC_SIZE");
  if (value) {
    long long unsigned int size_env = utils::parse_nvshmem_symmetric_size(value);
    if (size_env < required_symmetric_heap_size) {
      fprintf(stderr, "ERROR: Required > Current NVSHMEM_SYMMETRIC_SIZE=%s\n", value);
      MPI_CALL(MPI_Finalize());
      exit(1);
    }
  } else {
    char symmetric_heap_size_str[100];
    sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
    if (rank == 0)
      printf("Setting environment variable NVSHMEM_SYMMETRIC_SIZE = %llu\n", required_symmetric_heap_size);
    setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
  }
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
#else 
  nvshmem_init();
  int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  CUDA_SAFE_CALL(hipSetDevice(mype_node));
  //hipStream_t stream;
  //hipStreamCreate(&stream);
#endif

  int npes = nvshmem_n_pes();
  int mype = nvshmem_my_pe();
#ifdef USE_MPI
  assert(mype == rank);
#else
  auto &subg = *pg.get_subgraph(mype);
#endif
  nvshmem_barrier_all();
  //std::cout << "npes = " << npes << ", mype = " << mype << "\n";

  Timer t;
  t.Start();
  GraphGPU d_graph(nv, max_subg_ne, 0, 0, 0, 0, ndevices, 1);
  d_graph.allocate_nvshmem(nv, max_subg_ne, md, mype);
  std::cout << "PE[" << mype << "] subgraph[" << mype << "] has " << subg.V() << " vertices and " << subg.E() << " edges\n";
  d_graph.init_nvshmem(subg, mype);
  t.Stop();
  std::cout << "PE[" << mype << "] Total time allocating nvshmem and copying subgraphs to GPUs: " << t.Seconds() <<  " sec\n";
 
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_vertex_nvshmem, nthreads, 0);
  //std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(6*max_blocks, nblocks); 
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  size_t nwarps = WARPS_PER_BLOCK;
  size_t per_block_buffer_size = nwarps * size_t(md) * sizeof(vidType);
  size_t buffer_size = nblocks * per_block_buffer_size;
  std::cout << "frontier list size: " << float(buffer_size)/float(1024*1024) << " MB\n";
  vidType *buffers; // each warp has (k-3) vertex sets; each set has size of max_degree
  CUDA_SAFE_CALL(hipMalloc((void **)&buffers, buffer_size));
  nvshmem_barrier_all();

  double start_time = MPI_Wtime();
  t.Start();
  AccType h_count = 0;
  AccType * d_count = (AccType *)nvshmem_malloc(sizeof(AccType));
  CUDA_SAFE_CALL(hipMemcpy(d_count, &h_count, sizeof(AccType), hipMemcpyHostToDevice));
  vidType begin = mype * subgraph_size;
  vidType end = (mype+1) * subgraph_size;
  if (end > nv) end = nv;
  std::cout << "PE[" << mype << "] Start kernel: begin " << begin << " end " << end << " \n";
  warp_vertex_nvshmem<<<nblocks, nthreads>>>(begin, end, d_graph, buffers, mype, ndevices, md, d_count);
  CUDA_SAFE_CALL(hipMemcpy(&h_count, d_count, sizeof(AccType), hipMemcpyDeviceToHost));
  t.Stop();
  MPI_Barrier(MPI_COMM_WORLD);
  double end_time = MPI_Wtime();
  nvshmem_barrier_all();
  if (rank == 0) std::cout << "Global runtime = " << end_time - start_time << " sec\n";
  std::cout << "runtime[gpu" << mype << "] = " << t.Seconds() <<  " sec\n";
  //std::cout << "num_triangles[gpu" << mype << "] = " << h_count << "\n";
#ifdef USE_MPI
  uint64_t global_count = 0, local_count = h_count;
  MPI_Allreduce(&local_count, &global_count, 1, MPI_UINT64_T, MPI_SUM, MPI_COMM_WORLD);
  //if (rank == 0) std::cout << "Total triangle count = " << global_count << "\n";
  total = global_count;
  MPI_Finalize();
#else 
  nvshmem_finalize();
#endif
}

